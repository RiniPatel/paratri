#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/pair.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "cycletimer.h"

#include <utility>

using namespace std;

#define MAX_TRIANGLES 20000000

#define LOG2_THREADS_PER_BLOCK 9
#define THREADS_PER_BLOCK (1U << LOG2_THREADS_PER_BLOCK)

#define VERTICE_PER_THREAD 1

struct vertice_t
{
    uint32_t vertexID;
    uint32_t * x_col_end;
};

/* Helper function to round up to a power of 2. 
 */
inline int nextPow2(int n)
{
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

__global__ void
aggregate_kernel(uint32_t N, uint32_t *device_count, uint32_t *triangle_list, uint32_t *device_output)
{
    // compute overall index from position of thread in current block,
    // and given the block we are in
    int idx = (blockIdx.x << LOG2_THREADS_PER_BLOCK) + threadIdx.x;
    
    uint32_t start = (idx == 0)? 0 : device_count[idx - 1];
    uint32_t end = device_count[idx];
    
    // call cuda kernel
    const int threadsPerBlock = THREADS_PER_BLOCK;
    const int verticePerBlock = THREADS_PER_BLOCK * VERTICE_PER_THREAD;
    const int blocks = (N + verticePerBlock - 1) / verticePerBlock;
    const int totalThreads = (threadsPerBlock * blocks);
    int thr_idx = idx * (MAX_TRIANGLES/totalThreads);

    for(int c = start; c < end; c++) {
        int oidx = 3 * c;
        int iidx = 3 * (c - start);
        device_output[oidx] = triangle_list[thr_idx + iidx];
        device_output[oidx + 1] = triangle_list[thr_idx + iidx + 1];
        device_output[oidx + 2] = triangle_list[thr_idx + iidx + 2];        
    }
}

__global__ void
triangle_kernel(uint32_t *IA, uint32_t *JA, uint32_t N, uint32_t NUM_A, uint32_t *device_count, uint32_t * device_foo, vertice_t * device_vertice, uint32_t *triangle_list)
{
    // compute overall index from position of thread in current block,
    // and given the block we are in
    int globalThreadIdx = (blockIdx.x << LOG2_THREADS_PER_BLOCK) + threadIdx.x;

    int beginVertex = globalThreadIdx * VERTICE_PER_THREAD;
    int endVertex = min(beginVertex + VERTICE_PER_THREAD, N - 1);
    
    beginVertex = (beginVertex == 0)? beginVertex + 1: beginVertex;
    int bar = 0;

    for (int i = beginVertex; i < endVertex; i++)
    {
        uint32_t num_nnz_curr_row_x = IA[i + 1] - IA[i];
        uint32_t *x_col_begin = &JA[IA[i]];
        uint32_t *row_bound = &JA[IA[i + 1]];
        uint32_t *x_col_end = row_bound;
        

        for (uint32_t idx = 0; idx < num_nnz_curr_row_x; idx++)
        {
            if (x_col_begin[idx] > (i - 1)) {
                x_col_end = &x_col_begin[idx];
                break;
            }
        }

        uint32_t num_nnz_y = (row_bound - x_col_end);
        uint32_t num_nnz_x = (x_col_end - x_col_begin);
        bar += (num_nnz_x * num_nnz_y);

        device_foo[globalThreadIdx] = bar;
        device_vertice[globalThreadIdx].vertexID = beginVertex;
        device_vertice[globalThreadIdx].x_col_end = x_col_end;
    }

    
}

__global__ void triangle_kernel2(uint32_t *IA, uint32_t *JA, uint32_t N, uint32_t NUM_A, uint32_t *device_count, uint32_t * device_foo, vertice_t * device_vertice, uint32_t *triangle_list)
{
     // compute overall index from position of thread in current block,
    // and given the block we are in
    int globalThreadIdx = (blockIdx.x << LOG2_THREADS_PER_BLOCK) + threadIdx.x;

    // call cuda kernel
    const int threadsPerBlock = THREADS_PER_BLOCK;
    const int verticePerBlock = THREADS_PER_BLOCK * VERTICE_PER_THREAD;
    const int blocks = (N + verticePerBlock - 1) / verticePerBlock;
    const int totalThreads = (threadsPerBlock * blocks);

    int beginVertex = globalThreadIdx * VERTICE_PER_THREAD;
    int endVertex = min(beginVertex + VERTICE_PER_THREAD, N - 1);
    
    beginVertex = (beginVertex == 0)? beginVertex + 1: beginVertex;

    int delta = 0;

    for (int bla = beginVertex; bla < endVertex; bla++)
    {
        int i = device_vertice[bla].vertexID;
        if (device_foo[bla] == 0)
            return;
        // int i = bla;
        uint32_t num_nnz_curr_row_x = IA[i + 1] - IA[i];
        uint32_t *x_col_begin = &JA[IA[i]];
        uint32_t *row_bound = &JA[IA[i + 1]];
        uint32_t *x_col_end = row_bound;
        

        for (uint32_t idx = 0; idx < num_nnz_curr_row_x; idx++)
        {
            if (x_col_begin[idx] > (i - 1)) {
                x_col_end = &x_col_begin[idx];
                break;
            }
        }

        uint32_t num_nnz_y = (row_bound - x_col_end);
        uint32_t num_nnz_x = (x_col_end - x_col_begin);

        // This is where the real triangle counting begins.
        // We search through all possible vertices for x
        for (uint32_t j = 0; j < num_nnz_y; ++j) {
            uint32_t *x_col = x_col_begin;
            uint32_t *A_col = JA + IA[x_col_end[j]];
            uint32_t *A_col_max = JA + IA[x_col_end[j] + 1];

            // this loop searches through all possible vertices for z.
            for (uint32_t k = 0; k < num_nnz_x; ++k) {
                while ((*A_col < x_col[k])  && (A_col < A_col_max)) ++A_col;

                // for triangle enumeration i, *x_col_end, *A_col
                if (*A_col == x_col[k]) {
                    int idx = delta * 3;
                    uint32_t* ptr = (uint32_t*)(triangle_list + globalThreadIdx * (MAX_TRIANGLES/totalThreads));
                    ptr[idx] = i;
                    ptr[idx+1] = *A_col;
                    ptr[idx+2] = x_col_end[j];                
                    delta++;
                }
            }
        }
    }

    device_count[globalThreadIdx] = delta;
}

uint32_t count_triangles_cuda(uint32_t *IA, uint32_t *JA, uint32_t N, uint32_t NUM_A, uint32_t * output)
{
    uint32_t num_triangles = 0;
    
    uint32_t *device_IA, *device_JA;
    uint32_t *device_count, *triangle_list;
    uint32_t *device_output;

    uint32_t *device_foo;
    vertice_t *device_vertice;


    

    // call cuda kernel
    const int threadsPerBlock = THREADS_PER_BLOCK;
    const int verticePerBlock = THREADS_PER_BLOCK * VERTICE_PER_THREAD;
    const int blocks = (N + verticePerBlock - 1) / verticePerBlock;
    const int totalThreads = (threadsPerBlock * blocks);

    hipMalloc((void **)&device_IA, (N + 1) * sizeof(uint32_t));
    hipMalloc((void **)&device_JA, NUM_A * sizeof(uint32_t));
    hipMemcpy(device_IA, IA, (N + 1) * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(device_JA, JA, NUM_A * sizeof(uint32_t), hipMemcpyHostToDevice);
    uint32_t * host_foo = (uint32_t *) malloc(sizeof(uint32_t) * totalThreads);
    vertice_t * host_vertice = (vertice_t *) malloc(sizeof(vertice_t) * totalThreads);
    
    hipMalloc((void **)&triangle_list, MAX_TRIANGLES * sizeof(uint32_t) * 3);
    hipMalloc((void **)&device_count, totalThreads * sizeof(uint32_t));
    hipMalloc((void **)&device_foo, totalThreads * sizeof(uint32_t));
    hipMalloc((void **)&device_vertice, totalThreads * sizeof(vertice_t));    
    hipMalloc((void **)&device_output, MAX_TRIANGLES * sizeof(uint32_t) * 3);

    double start = currentSeconds();

    triangle_kernel<<<blocks, threadsPerBlock>>>(device_IA, device_JA, N, NUM_A, device_count, device_foo, device_vertice, triangle_list);
    hipDeviceSynchronize();

    // hipMemcpy(host_foo, device_foo, totalThreads * sizeof(uint32_t), hipMemcpyDeviceToHost);    
    // hipMemcpy(host_vertice, device_vertice, totalThreads * sizeof(vertice_t), hipMemcpyDeviceToHost);

    // for (uint64_t i = 0; i < totalThreads; i++)
    // {
    //     printf("%lu,%lu\n", host_foo[i], host_vertice[i].vertexID);
    // }

    thrust::device_ptr<uint32_t> device_foo_thrust = thrust::device_pointer_cast(device_foo);
    thrust::device_ptr<vertice_t> device_vertice_thrust = thrust::device_pointer_cast(device_vertice);
    thrust::sort_by_key(device_foo_thrust, device_foo_thrust + totalThreads, device_vertice_thrust);

    triangle_kernel2<<<blocks, threadsPerBlock>>>(device_IA, device_JA, N, NUM_A, device_count, device_foo, device_vertice, triangle_list);

    thrust::device_ptr<uint32_t> device_count_thrust = thrust::device_pointer_cast(device_count);
    thrust::inclusive_scan(device_count_thrust, device_count_thrust + totalThreads, device_count_thrust);

    // aggregate_kernel<<<blocks, threadsPerBlock>>>(N, device_count, triangle_list, device_output);

    double end = currentSeconds();

    printf("CUDA computation time is %lf\n", end - start);

    hipMemcpy(&num_triangles, &device_count[totalThreads - 1], sizeof(uint32_t),
               hipMemcpyDeviceToHost);
    // hipMemcpy(triangle_list, device_output, sizeof(uint32_t) * num_triangles * 3,
               // hipMemcpyDeviceToHost);

    hipFree(device_IA);
    hipFree(device_JA);
    hipFree(device_count);
    hipFree(device_output);
    hipFree(triangle_list);

    return num_triangles;
}

void printCudaInfo()
{
    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
